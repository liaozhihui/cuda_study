//
// Created by 廖智慧 on 29.11.22.
//

#include <hip/hip_runtime.h>
#include <stdio.h>
__global__ void hello_from_gpu(){
    printf("Hello World from the GPU!\n");
}
int main(){
    hello_from_gpu<<<1,1>>>();
    hipDeviceSynchronize(); //同步主机与设备，刷新缓存区，输出字符
    return 0;
}
